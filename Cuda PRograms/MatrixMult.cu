
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <time.h>

void init_matrix(int *d, int N)
{

    printf("\n");
    for (int i = 0; i < N; i++)
    {
        d[i] = (rand() % 10);
    }
}

__global__ void mult_matrix(int *a, int *b, int *c)
{

    int global_index = threadIdx.x + blockDim.x * threadIdx.y;
    c[global_index] = a[global_index] + b[global_index];
}

__global__ void kwait(unsigned long long duration){
    unsigned long long start=clock64();
    while(clock64()< start + duration);
}

int main(int argc, char **argv)
{
    FILE *fptr = fopen("results.txt", "w");
    if (fptr == NULL)
    {
        printf("Error opening file my g");
        exit(1);
    }
    fprintf(fptr, "Spin Method: Duration :Size ");

    int N = 50;
    for (int i = 1; i < argc; i++)
    {
        if (strcmp(argv[i], "-size") == 0 && i + 1 < argc)
        {
            N = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-sync") == 0 && i + 1 < argc) {
            if (strcmp(argv[i + 1], "spin") == 0)
            {
                hipSetDeviceFlags(hipDeviceScheduleSpin);
                fprintf(fptr, "spin: \n");
            }
            else if (strcmp(argv[i + 1], "block") == 0)
            {
                hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
                fprintf(fptr, "block: \n");
            }
            else
            {
                printf("\n INVALID SYNC");
            }
        }
    }

    for (int i = 10; i <= N; i += 10)
    {

        clock_t start, end;
        double duration;

        int kernelLaunches; // for input later

        srand(time(NULL));

        for (int j = 1; j < argc; j++)
        {

            if (strcmp(argv[j], "-n") == 0 && j+ 1 < argc)
            {
                kernelLaunches = atoi(argv[j + 1]);
            }
           
        }
        size_t bytes = i * i * sizeof(int);
        int *a, *b, *c;

        hipMallocManaged(&a, bytes);
        hipMallocManaged(&b, bytes);
        hipMallocManaged(&c, bytes);

        int threads = 16;
        int blocks = (i + threads - 1) / threads;

        dim3 THREADS(threads, threads);
        dim3 BLOCKS(blocks, blocks);

        init_matrix(a, i);
        init_matrix(b, i);
        const unsigned long long my_duration= 2000000000ULL;
        start = clock();
        for (int j = 0; j <= kernelLaunches; j++)
        {
           // mult_matrix<<<BLOCKS, THREADS>>>(a, b, c);
            kwait<<<1,1>>>(my_duration);
            int rc=hipDeviceSynchronize();
            printf("Size: %d During Running %d + %d = %d| rc %d \n",i, a[i-1], b[i-1], c[i-1],rc);
        }

        end = clock();

        duration = ((double)(end - start)) / CLOCKS_PER_SEC;
        printf("Total Duration: %f \n", duration);
        fprintf(fptr, "%f :", duration);

        double avgDuration = duration / kernelLaunches;

        printf("Average time for each kernel: %f", avgDuration);
        fprintf(fptr, " %f :", avgDuration);

        printf("Size %d\n", i);
        fprintf(fptr, "%d\n", i);

        printf("\n %d + %d = %d \n", a[i], b[i], c[i]);

        hipFree(a);
        hipFree(b);
        hipFree(c);
    }
}
